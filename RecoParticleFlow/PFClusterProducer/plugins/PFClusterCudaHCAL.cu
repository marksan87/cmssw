#include "hip/hip_runtime.h"

#include <cmath>
#include <iostream>

// CUDA include files
#include <hip/hip_runtime.h>

// CMSSW include files
#include "HeterogeneousCore/CUDAUtilities/interface/cudaCheck.h"
#include "RecoParticleFlow/PFClusterProducer/plugins/PFClusterCudaHCAL.h"
#include <Eigen/Dense>

namespace PFClusterCudaHCAL {

  // THE ART OF HARDCODING
  // these numbers should be copied over during initialization
  __constant__ float showerSigma = 10;
  
  __constant__ float recHitEnergyNormEB_1 = 0.1;
  __constant__ float recHitEnergyNormEB_2 = 0.2;
  __constant__ float recHitEnergyNormEB_3 = 0.3;
  __constant__ float recHitEnergyNormEB_4 = 0.3;
  __constant__ float recHitEnergyNormEE_1 = 0.1;
  __constant__ float recHitEnergyNormEE_2_7 = 0.2;
  
  __constant__ float minFracToKeep = 0.0000001;

  __constant__ float seedEThresholdEB_1 = 0.125;
  __constant__ float seedEThresholdEB_2 = 0.25;
  __constant__ float seedEThresholdEB_3 = 0.35;
  __constant__ float seedEThresholdEB_4 = 0.35;
  __constant__ float seedEThresholdEE_1 = 0.1375;
  __constant__ float seedEThresholdEE_2_7 = 0.275;

  __constant__ float seedPt2ThresholdEB = 0.;
  __constant__ float seedPt2hresholdEE = 0.;

  __constant__ float topoEThresholdEB_1 = 0.1;
  __constant__ float topoEThresholdEB_2 = 0.2;
  __constant__ float topoEThresholdEB_3 = 0.3;
  __constant__ float topoEThresholdEB_4 = 0.3;
  __constant__ float topoEThresholdEE_1 = 0.1;
  __constant__ float topoEThresholdEE_2_7 = 0.2;
  __constant__ float topoEThresholdEB_vec[4] = {0.1,0.2,0.3,0.3};
  __constant__ float topoEThresholdEE_vec[7] = {0.1,0.2,0.2,0.2,0.2,0.2,0.2};
  

  __constant__ int nNeighTopo = 8;
  __constant__ int nNT = 8;
  __constant__ int nNeigh = 4;
  __constant__ int maxSize = 100;
  
  //int nTopoLoops = 100; // 35;
  int nTopoLoops = 35;


 __global__ void seedingKernel_HCAL(
     				    size_t size, 
				    double* pfrh_energy,
				    double* pfrh_pt2,
				    int*   pfrh_isSeed,
				    int*   pfrh_topoId,
				    int*   pfrh_layer,
				    int*   pfrh_depth,
				    int*   neigh4_Ind
				    ) {

   int i = threadIdx.x+blockIdx.x*blockDim.x;

   if(i<size) {        
     if( ( pfrh_layer[i] == 1 && 
	   pfrh_depth[i] == 1 &&
	   pfrh_energy[i]>seedEThresholdEB_1 && 
	   pfrh_pt2[i]>seedPt2ThresholdEB ) ||
	 ( pfrh_layer[i] == 1 && 
	   pfrh_depth[i] == 2 &&
	   pfrh_energy[i]>seedEThresholdEB_2 && 
	   pfrh_pt2[i]>seedPt2ThresholdEB ) ||
	 ( pfrh_layer[i] == 1 && 
	   pfrh_depth[i] == 3 &&
	   pfrh_energy[i]>seedEThresholdEB_3 && 
	   pfrh_pt2[i]>seedPt2ThresholdEB ) ||
	 ( pfrh_layer[i] == 1 && 
	   pfrh_depth[i] == 4 &&
	   pfrh_energy[i]>seedEThresholdEB_4 && 
	   pfrh_pt2[i]>seedPt2ThresholdEB ) ||
	 ( pfrh_layer[i] == 3  && 
	   pfrh_depth[i] == 1  &&
	   pfrh_energy[i]>seedEThresholdEE_1 && 
	   pfrh_pt2[i]>seedPt2hresholdEE)   ||
	 ( pfrh_layer[i] == 3  && 
	   pfrh_depth[i] > 1   &&
	   pfrh_energy[i]>seedEThresholdEE_2_7 && 
	   pfrh_pt2[i]>seedPt2hresholdEE))
       {
	 pfrh_isSeed[i]=1;		 
	 for(int k=0; k<nNeigh; k++){
	   if(neigh4_Ind[nNeigh*i+k]<0) continue; 
	   if(pfrh_energy[i]<pfrh_energy[neigh4_Ind[nNeigh*i+k]]){
	     pfrh_isSeed[i]=0;
	     //pfrh_topoId[i]=-1;	     
	     break;
	   }
	 }		
       }
     else{ 
       // pfrh_topoId[i]=-1;
       pfrh_isSeed[i]=0;
       	    
     }     
   }
 }
  
  
 __global__ void seedingKernel_HCAL_serialize(
     				    size_t size, 
				    double* pfrh_energy,
				    double* pfrh_pt2,
				    int*   pfrh_isSeed,
				    int*   pfrh_topoId,
				    int*   pfrh_layer,
				    int*   pfrh_depth,
				    int*   neigh4_Ind
				    ) {

   //int i = threadIdx.x+blockIdx.x*blockDim.x;
   for (int i = 0; i < size; i++) {
       if(i<size) {        
         if( ( pfrh_layer[i] == 1 && 
           pfrh_depth[i] == 1 &&
           pfrh_energy[i]>seedEThresholdEB_1 && 
           pfrh_pt2[i]>seedPt2ThresholdEB ) ||
         ( pfrh_layer[i] == 1 && 
           pfrh_depth[i] == 2 &&
           pfrh_energy[i]>seedEThresholdEB_2 && 
           pfrh_pt2[i]>seedPt2ThresholdEB ) ||
         ( pfrh_layer[i] == 1 && 
           pfrh_depth[i] == 3 &&
           pfrh_energy[i]>seedEThresholdEB_3 && 
           pfrh_pt2[i]>seedPt2ThresholdEB ) ||
         ( pfrh_layer[i] == 1 && 
           pfrh_depth[i] == 4 &&
           pfrh_energy[i]>seedEThresholdEB_4 && 
           pfrh_pt2[i]>seedPt2ThresholdEB ) ||
         ( pfrh_layer[i] == 3  && 
           pfrh_depth[i] == 1  &&
           pfrh_energy[i]>seedEThresholdEE_1 && 
           pfrh_pt2[i]>seedPt2hresholdEE)   ||
         ( pfrh_layer[i] == 3  && 
           pfrh_depth[i] > 1   &&
           pfrh_energy[i]>seedEThresholdEE_2_7 && 
           pfrh_pt2[i]>seedPt2hresholdEE))
           {
         pfrh_isSeed[i]=1;		 
         for(int k=0; k<nNeigh; k++){
           if(neigh4_Ind[nNeigh*i+k]<0) continue; 
           if(pfrh_energy[i]<pfrh_energy[neigh4_Ind[nNeigh*i+k]]){
             pfrh_isSeed[i]=0;
             //pfrh_topoId[i]=-1;	     
             break;
           }
         }		
           }
         else{ 
           // pfrh_topoId[i]=-1;
           pfrh_isSeed[i]=0;
                
         }     
       }
    }
 }
  

   __global__ void topoKernel_HCALV2( 
				  size_t size,
				  double* pfrh_energy,
				  int* pfrh_topoId,
				  int* pfrh_layer,
				  int* pfrh_depth,
				  int* neigh8_Ind
				  ) {
     
     int l = threadIdx.x+blockIdx.x*blockDim.x;
     //int k = threadIdx.y+blockIdx.y*blockDim.y;
     int k = (threadIdx.y+blockIdx.y*blockDim.y) % nNT;
           
      //if(l<size && k<nNT) {
      if(l<size) {

	while( neigh8_Ind[nNT*l+k] > -1 && pfrh_topoId[l] != pfrh_topoId[neigh8_Ind[nNT*l+k]] && 
	    ( (pfrh_layer[neigh8_Ind[nNT*l+k]] == 3 && pfrh_energy[neigh8_Ind[nNT*l+k]]>topoEThresholdEE_vec[pfrh_depth[neigh8_Ind[nNT*l+k]]-1]) ||
	      (pfrh_layer[neigh8_Ind[nNT*l+k]] == 1 && pfrh_energy[neigh8_Ind[nNT*l+k]]>topoEThresholdEB_vec[pfrh_depth[neigh8_Ind[nNT*l+k]]-1]) ) &&
	    ( (pfrh_layer[l] == 3 && pfrh_energy[l]>topoEThresholdEE_vec[pfrh_depth[l]-1]) ||
	      (pfrh_layer[l] == 1 && pfrh_energy[l]>topoEThresholdEB_vec[pfrh_depth[l]-1]))
	    )
	    {
	      if(pfrh_topoId[l] > pfrh_topoId[neigh8_Ind[nNT*l+k]]){
		atomicMax(&pfrh_topoId[neigh8_Ind[nNT*l+k]],pfrh_topoId[l]);
	      }
	      if(pfrh_topoId[l] < pfrh_topoId[neigh8_Ind[nNT*l+k]]){
		atomicMax(&pfrh_topoId[l], pfrh_topoId[neigh8_Ind[nNT*l+k]]);
	      }
	    }	
      }
   }
 

   __global__ void topoKernel_HCAL_serialize( 
				  size_t size,
				  double* pfrh_energy,
				  int* pfrh_topoId,
				  int* pfrh_layer,
				  int* pfrh_depth,
				  int* neigh8_Ind
				  ) {
     
     //int l = threadIdx.x+blockIdx.x*blockDim.x;
     //int k = threadIdx.y+blockIdx.y*blockDim.y;
     
     for (int l = 0; l < size; l++) {
        //for (int k = 0; k < size; k++) {
        for (int k = 0; k < 8; k++) {
           
            while( neigh8_Ind[nNT*l+k] > -1 && pfrh_topoId[l] != pfrh_topoId[neigh8_Ind[nNT*l+k]] && 
                ( (pfrh_layer[neigh8_Ind[nNT*l+k]] == 3 && pfrh_energy[neigh8_Ind[nNT*l+k]]>topoEThresholdEE_vec[pfrh_depth[neigh8_Ind[nNT*l+k]]-1]) ||
                  (pfrh_layer[neigh8_Ind[nNT*l+k]] == 1 && pfrh_energy[neigh8_Ind[nNT*l+k]]>topoEThresholdEB_vec[pfrh_depth[neigh8_Ind[nNT*l+k]]-1]) ) &&
                ( (pfrh_layer[l] == 3 && pfrh_energy[l]>topoEThresholdEE_vec[pfrh_depth[l]-1]) ||
                  (pfrh_layer[l] == 1 && pfrh_energy[l]>topoEThresholdEB_vec[pfrh_depth[l]-1]))
                )
                {
                  if(pfrh_topoId[l] > pfrh_topoId[neigh8_Ind[nNT*l+k]]){
                atomicMax(&pfrh_topoId[neigh8_Ind[nNT*l+k]],pfrh_topoId[l]);
                  }
                  if(pfrh_topoId[l] < pfrh_topoId[neigh8_Ind[nNT*l+k]]){
                atomicMax(&pfrh_topoId[l], pfrh_topoId[neigh8_Ind[nNT*l+k]]);
                  }
                }	
          }
        }
   }
 
__global__ void hcalFastCluster_step1( size_t size,
					     float* pfrh_x,
					     float* pfrh_y,
					     float* pfrh_z,
					     double* pfrh_energy,
					     int* pfrh_topoId,
					     int* pfrh_isSeed,
					     int* pfrh_layer,
				             int* pfrh_depth,
					     float* pcrhfrac,
					     int* pcrhfracind,
					     float* fracSum,
					     int* rhCount
					     ) {

    int i = threadIdx.x+blockIdx.x*blockDim.x;
    int j = threadIdx.y+blockIdx.y*blockDim.y;
    //make sure topoID, Layer is the same, i is seed and j is not seed
    if( i<size && j<size){

      if( pfrh_topoId[i] == pfrh_topoId[j] && pfrh_isSeed[i]==1 ){

      float dist2 =
	       (pfrh_x[i] - pfrh_x[j])*(pfrh_x[i] - pfrh_x[j])
	      +(pfrh_y[i] - pfrh_y[j])*(pfrh_y[i] - pfrh_y[j])
	      +(pfrh_z[i] - pfrh_z[j])*(pfrh_z[i] - pfrh_z[j]);

      float d2 = dist2 / (showerSigma*showerSigma);
      float fraction = -1.;

      if(pfrh_layer[j] == 1 && pfrh_depth[j] == 1) { fraction = pfrh_energy[i] / recHitEnergyNormEB_1 * expf(-0.5 * d2); }
      if(pfrh_layer[j] == 1 && pfrh_depth[j] == 2) { fraction = pfrh_energy[i] / recHitEnergyNormEB_2 * expf(-0.5 * d2); }
      if(pfrh_layer[j] == 1 && pfrh_depth[j] == 3) { fraction = pfrh_energy[i] / recHitEnergyNormEB_3 * expf(-0.5 * d2); }
      if(pfrh_layer[j] == 1 && pfrh_depth[j] == 4) { fraction = pfrh_energy[i] / recHitEnergyNormEB_4 * expf(-0.5 * d2); }
      if(pfrh_layer[j] == 3 && pfrh_depth[j] == 1) { fraction = pfrh_energy[i] / recHitEnergyNormEE_1 * expf(-0.5 * d2); }
      if(pfrh_layer[j] == 3 && pfrh_depth[j] > 1 ) { fraction = pfrh_energy[i] / recHitEnergyNormEE_2_7 * expf(-0.5 * d2); }
	  
      if(fraction==-1.) printf("FRACTION is NEGATIVE!!!");

      if( pfrh_isSeed[j]!=1 && d2<100.)
	{
	  atomicAdd(&fracSum[j],fraction);
	}
      }
    }
  }

 
__global__ void hcalFastCluster_step1_serialize( size_t size,
					     float* pfrh_x,
					     float* pfrh_y,
					     float* pfrh_z,
					     double* pfrh_energy,
					     int* pfrh_topoId,
					     int* pfrh_isSeed,
					     int* pfrh_layer,
				             int* pfrh_depth,
					     float* pcrhfrac,
					     int* pcrhfracind,
					     float* fracSum,
					     int* rhCount
					     ) {

    //int i = threadIdx.x+blockIdx.x*blockDim.x;
    //int j = threadIdx.y+blockIdx.y*blockDim.y;
    for (int i = 0; i < size; i++) {
        for (int j = 0; j < size; j++) {
            //make sure topoID, Layer is the same, i is seed and j is not seed
            if( i<size && j<size){

              if( pfrh_topoId[i] == pfrh_topoId[j] && pfrh_isSeed[i]==1 ){

              float dist2 =
                   (pfrh_x[i] - pfrh_x[j])*(pfrh_x[i] - pfrh_x[j])
                  +(pfrh_y[i] - pfrh_y[j])*(pfrh_y[i] - pfrh_y[j])
                  +(pfrh_z[i] - pfrh_z[j])*(pfrh_z[i] - pfrh_z[j]);

              float d2 = dist2 / (showerSigma*showerSigma);
              float fraction = -1.;

              if(pfrh_layer[j] == 1 && pfrh_depth[j] == 1) { fraction = pfrh_energy[i] / recHitEnergyNormEB_1 * expf(-0.5 * d2); }
              if(pfrh_layer[j] == 1 && pfrh_depth[j] == 2) { fraction = pfrh_energy[i] / recHitEnergyNormEB_2 * expf(-0.5 * d2); }
              if(pfrh_layer[j] == 1 && pfrh_depth[j] == 3) { fraction = pfrh_energy[i] / recHitEnergyNormEB_3 * expf(-0.5 * d2); }
              if(pfrh_layer[j] == 1 && pfrh_depth[j] == 4) { fraction = pfrh_energy[i] / recHitEnergyNormEB_4 * expf(-0.5 * d2); }
              if(pfrh_layer[j] == 3 && pfrh_depth[j] == 1) { fraction = pfrh_energy[i] / recHitEnergyNormEE_1 * expf(-0.5 * d2); }
              if(pfrh_layer[j] == 3 && pfrh_depth[j] > 1 ) { fraction = pfrh_energy[i] / recHitEnergyNormEE_2_7 * expf(-0.5 * d2); }
              
              if(fraction==-1.) printf("FRACTION is NEGATIVE!!!");

              if( pfrh_isSeed[j]!=1 && d2<100.)
            {
              atomicAdd(&fracSum[j],fraction);
            }
              }
            }
        }
    }
  }

__global__ void hcalFastCluster_step2( size_t size,
					     float* pfrh_x,
					     float* pfrh_y,
					     float* pfrh_z,
					     double* pfrh_energy,
					     int* pfrh_topoId,
					     int* pfrh_isSeed,
					     int* pfrh_layer,
				             int* pfrh_depth,
					     float* pcrhfrac,
					     int* pcrhfracind,
					     float* fracSum,
					     int* rhCount
					     ) {

    int i = threadIdx.x+blockIdx.x*blockDim.x;
    int j = threadIdx.y+blockIdx.y*blockDim.y;
    //make sure topoID, Layer is the same, i is seed and j is not seed
    if( i<size && j<size){
      if( pfrh_topoId[i] == pfrh_topoId[j] && pfrh_isSeed[i]==1 ){
      if(i==j)
	{
	  pcrhfrac[i*maxSize]    = 1.;
	  pcrhfracind[i*maxSize] = j;
	}
      if( pfrh_isSeed[j]!=1 ){
	float dist2 =
	   (pfrh_x[i] - pfrh_x[j])*(pfrh_x[i] - pfrh_x[j])
	  +(pfrh_y[i] - pfrh_y[j])*(pfrh_y[i] - pfrh_y[j])
	  +(pfrh_z[i] - pfrh_z[j])*(pfrh_z[i] - pfrh_z[j]);

	float d2 = dist2 / (showerSigma*showerSigma);
	float fraction = -1.;

	if(pfrh_layer[j] == 1 && pfrh_depth[j] == 1) { fraction = pfrh_energy[i] / recHitEnergyNormEB_1 * expf(-0.5 * d2); }
	if(pfrh_layer[j] == 1 && pfrh_depth[j] == 2) { fraction = pfrh_energy[i] / recHitEnergyNormEB_2 * expf(-0.5 * d2); }
	if(pfrh_layer[j] == 1 && pfrh_depth[j] == 3) { fraction = pfrh_energy[i] / recHitEnergyNormEB_3 * expf(-0.5 * d2); }
	if(pfrh_layer[j] == 1 && pfrh_depth[j] == 4) { fraction = pfrh_energy[i] / recHitEnergyNormEB_4 * expf(-0.5 * d2); }
	if(pfrh_layer[j] == 3 && pfrh_depth[j] == 1) { fraction = pfrh_energy[i] / recHitEnergyNormEE_1 * expf(-0.5 * d2); }
	if(pfrh_layer[j] == 3 && pfrh_depth[j] > 1 ) { fraction = pfrh_energy[i] / recHitEnergyNormEE_2_7 * expf(-0.5 * d2); }
	  
	
	if(fraction==-1.) printf("FRACTION is NEGATIVE!!!");
	if(d2 < 100. )
	  {
	    if ((fraction/fracSum[j])>minFracToKeep){
	      int k = atomicAdd(&rhCount[i],1);
	      pcrhfrac[i*maxSize+k] = fraction/fracSum[j];
	      pcrhfracind[i*maxSize+k] = j;
	      //printf("(i,j)=(%d,%d), rhCount=%d, fraction=%f, fracsum=%f \n",i,j,rhCount[i], fraction, fracSum[j]);
	    }
	  }
      }
      }
    }
}


__global__ void hcalFastCluster_step2_serialize( size_t size,
					     float* pfrh_x,
					     float* pfrh_y,
					     float* pfrh_z,
					     double* pfrh_energy,
					     int* pfrh_topoId,
					     int* pfrh_isSeed,
					     int* pfrh_layer,
				             int* pfrh_depth,
					     float* pcrhfrac,
					     int* pcrhfracind,
					     float* fracSum,
					     int* rhCount
					     ) {

    //int i = threadIdx.x+blockIdx.x*blockDim.x;
    //int j = threadIdx.y+blockIdx.y*blockDim.y;
    //make sure topoID, Layer is the same, i is seed and j is not seed
    for (int i = 0; i < size; i++) {
        for (int j = 0; j < size; j++) {
            if( i<size && j<size){
              if( pfrh_topoId[i] == pfrh_topoId[j] && pfrh_isSeed[i]==1 ){
              if(i==j)
            {
              pcrhfrac[i*maxSize]    = 1.;
              pcrhfracind[i*maxSize] = j;
            }
              if( pfrh_isSeed[j]!=1 ){
            float dist2 =
               (pfrh_x[i] - pfrh_x[j])*(pfrh_x[i] - pfrh_x[j])
              +(pfrh_y[i] - pfrh_y[j])*(pfrh_y[i] - pfrh_y[j])
              +(pfrh_z[i] - pfrh_z[j])*(pfrh_z[i] - pfrh_z[j]);

            float d2 = dist2 / (showerSigma*showerSigma);
            float fraction = -1.;

            if(pfrh_layer[j] == 1 && pfrh_depth[j] == 1) { fraction = pfrh_energy[i] / recHitEnergyNormEB_1 * expf(-0.5 * d2); }
            if(pfrh_layer[j] == 1 && pfrh_depth[j] == 2) { fraction = pfrh_energy[i] / recHitEnergyNormEB_2 * expf(-0.5 * d2); }
            if(pfrh_layer[j] == 1 && pfrh_depth[j] == 3) { fraction = pfrh_energy[i] / recHitEnergyNormEB_3 * expf(-0.5 * d2); }
            if(pfrh_layer[j] == 1 && pfrh_depth[j] == 4) { fraction = pfrh_energy[i] / recHitEnergyNormEB_4 * expf(-0.5 * d2); }
            if(pfrh_layer[j] == 3 && pfrh_depth[j] == 1) { fraction = pfrh_energy[i] / recHitEnergyNormEE_1 * expf(-0.5 * d2); }
            if(pfrh_layer[j] == 3 && pfrh_depth[j] > 1 ) { fraction = pfrh_energy[i] / recHitEnergyNormEE_2_7 * expf(-0.5 * d2); }
              
            
            if(fraction==-1.) printf("FRACTION is NEGATIVE!!!");
            if(d2 < 100. )
              {
                if ((fraction/fracSum[j])>minFracToKeep){
                  int k = atomicAdd(&rhCount[i],1);
                  pcrhfrac[i*maxSize+k] = fraction/fracSum[j];
                  pcrhfracind[i*maxSize+k] = j;
                  //printf("(i,j)=(%d,%d), rhCount=%d, fraction=%f, fracsum=%f \n",i,j,rhCount[i], fraction, fracSum[j]);
                }
              }
              }
              }
            }
        }
    }
}



void PFRechitToPFCluster_HCALV2(size_t size, 
				float* pfrh_x, 
				float* pfrh_y, 
				float* pfrh_z, 
				double* pfrh_energy, 
				double* pfrh_pt2,    				
				int* pfrh_isSeed,
				int* pfrh_topoId, 
				int* pfrh_layer, 
				int* pfrh_depth, 
				int* neigh8_Ind, 
				int* neigh4_Ind, 				
				
				int* pcrhfracind,
				float* pcrhfrac,
				float* fracSum,
				int* rhCount,
				float* timer
                )
  {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    //seeding
    if(size>0) seedingKernel_HCAL<<<(size+512-1)/512, 512>>>( size,  pfrh_energy,   pfrh_pt2,   pfrh_isSeed,  pfrh_topoId,  pfrh_layer,pfrh_depth,  neigh4_Ind);
    //hipDeviceSynchronize();
    
    //topoclustering 
     
      //dim3 gridT( (size+64-1)/64, 1 );
      //dim3 blockT( 64, 8);
      dim3 gridT( (size+64-1)/64, 8 );
      dim3 blockT( 64, 16); // 16 threads in a half-warp
      hipEventRecord(start);
      for(int h=0;h<nTopoLoops; h++){
    
      if(size>0) topoKernel_HCALV2<<<gridT, blockT>>>( size, pfrh_energy,  pfrh_topoId,  pfrh_layer, pfrh_depth, neigh8_Ind);	     
      }
      //hipDeviceSynchronize();
   
      float milliseconds = 0;
      if (timer != nullptr)
      {
          hipEventRecord(stop);
          hipEventSynchronize(stop);   
          hipEventElapsedTime(&milliseconds, start, stop);
          *timer = milliseconds;
      }


      dim3 grid( (size+32-1)/32, (size+32-1)/32 );
      dim3 block( 32, 32);

      if(size>0) hcalFastCluster_step1<<<grid, block>>>( size, pfrh_x,  pfrh_y,  pfrh_z,  pfrh_energy, pfrh_topoId,  pfrh_isSeed,  pfrh_layer, pfrh_depth, pcrhfrac, pcrhfracind, fracSum, rhCount);
     //hipDeviceSynchronize();

      if(size>0) hcalFastCluster_step2<<<grid, block>>>( size, pfrh_x,  pfrh_y,  pfrh_z,  pfrh_energy, pfrh_topoId,  pfrh_isSeed,  pfrh_layer, pfrh_depth, pcrhfrac, pcrhfracind, fracSum, rhCount);


  }

void PFRechitToPFCluster_HCAL_serialize(size_t size, 
				float* pfrh_x, 
				float* pfrh_y, 
				float* pfrh_z, 
				double* pfrh_energy, 
				double* pfrh_pt2,    				
				int* pfrh_isSeed,
				int* pfrh_topoId, 
				int* pfrh_layer, 
				int* pfrh_depth, 
				int* neigh8_Ind, 
				int* neigh4_Ind, 				
				
				int* pcrhfracind,
				float* pcrhfrac,
				float* fracSum,
				int* rhCount,
				float* timer
				)
  {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    //seeding
    if(size>0) seedingKernel_HCAL_serialize<<<1,1>>>( size,  pfrh_energy,   pfrh_pt2,   pfrh_isSeed,  pfrh_topoId,  pfrh_layer,pfrh_depth,  neigh4_Ind);
    //hipDeviceSynchronize();
    
    //topoclustering 
      
      //dim3 gridT( (size+64-1)/64, 1 );
      //dim3 blockT( 64, 8);
      hipEventRecord(start);
      for(int h=0;h<nTopoLoops; h++){
    
      if(size>0) topoKernel_HCAL_serialize<<<1,1>>>( size, pfrh_energy,  pfrh_topoId,  pfrh_layer, pfrh_depth, neigh8_Ind);	     
      }
      //hipDeviceSynchronize();
      float milliseconds = 0;
      if (timer != nullptr)
      {
          hipEventRecord(stop);
          hipEventSynchronize(stop);
          hipEventElapsedTime(&milliseconds, start, stop);
          *timer = milliseconds;
      }
    

      //dim3 grid( (size+32-1)/32, (size+32-1)/32 );
      //dim3 block( 32, 32);

      if(size>0) hcalFastCluster_step1_serialize<<<1,1>>>( size, pfrh_x,  pfrh_y,  pfrh_z,  pfrh_energy, pfrh_topoId,  pfrh_isSeed,  pfrh_layer, pfrh_depth, pcrhfrac, pcrhfracind, fracSum, rhCount);
     //hipDeviceSynchronize();

      if(size>0) hcalFastCluster_step2_serialize<<<1,1>>>( size, pfrh_x,  pfrh_y,  pfrh_z,  pfrh_energy, pfrh_topoId,  pfrh_isSeed,  pfrh_layer, pfrh_depth, pcrhfrac, pcrhfracind, fracSum, rhCount);


  }

void PFRechitToPFCluster_HCAL_serialize_seedingParallel(size_t size, 
				float* pfrh_x, 
				float* pfrh_y, 
				float* pfrh_z, 
				double* pfrh_energy, 
				double* pfrh_pt2,    				
				int* pfrh_isSeed,
				int* pfrh_topoId, 
				int* pfrh_layer, 
				int* pfrh_depth, 
				int* neigh8_Ind, 
				int* neigh4_Ind, 				
				
				int* pcrhfracind,
				float* pcrhfrac,
				float* fracSum,
				int* rhCount
				)
  { 
    //seeding
    if(size>0) seedingKernel_HCAL<<<(size+512-1)/512, 512>>>( size,  pfrh_energy,   pfrh_pt2,   pfrh_isSeed,  pfrh_topoId,  pfrh_layer,pfrh_depth,  neigh4_Ind);
    //hipDeviceSynchronize();
    
    //topoclustering 
      
      //dim3 gridT( (size+64-1)/64, 1 );
      //dim3 blockT( 64, 8);
      for(int h=0;h<nTopoLoops; h++){
      if(size>0) topoKernel_HCAL_serialize<<<1,1>>>( size, pfrh_energy,  pfrh_topoId,  pfrh_layer, pfrh_depth, neigh8_Ind);
      }
     hipDeviceSynchronize();
    
    

      //dim3 grid( (size+32-1)/32, (size+32-1)/32 );
      //dim3 block( 32, 32);

      if(size>0) hcalFastCluster_step1_serialize<<<1,1>>>( size, pfrh_x,  pfrh_y,  pfrh_z,  pfrh_energy, pfrh_topoId,  pfrh_isSeed,  pfrh_layer, pfrh_depth, pcrhfrac, pcrhfracind, fracSum, rhCount);
     //hipDeviceSynchronize();

      if(size>0) hcalFastCluster_step2_serialize<<<1,1>>>( size, pfrh_x,  pfrh_y,  pfrh_z,  pfrh_energy, pfrh_topoId,  pfrh_isSeed,  pfrh_layer, pfrh_depth, pcrhfrac, pcrhfracind, fracSum, rhCount);


  }

void PFRechitToPFCluster_HCAL_serialize_topoParallel(size_t size, 
				float* pfrh_x, 
				float* pfrh_y, 
				float* pfrh_z, 
				double* pfrh_energy, 
				double* pfrh_pt2,    				
				int* pfrh_isSeed,
				int* pfrh_topoId, 
				int* pfrh_layer, 
				int* pfrh_depth, 
				int* neigh8_Ind, 
				int* neigh4_Ind, 				
				
				int* pcrhfracind,
				float* pcrhfrac,
				float* fracSum,
				int* rhCount
				)
  { 
    //seeding
    if(size>0) seedingKernel_HCAL_serialize<<<1,1>>>( size,  pfrh_energy,   pfrh_pt2,   pfrh_isSeed,  pfrh_topoId,  pfrh_layer,pfrh_depth,  neigh4_Ind);
    //hipDeviceSynchronize();
    
    //topoclustering 
      
      dim3 gridT( (size+64-1)/64, 1 );
      dim3 blockT( 64, 8);
      for(int h=0;h<nTopoLoops; h++){
   
      if(size>0) topoKernel_HCALV2<<<gridT, blockT>>>( size, pfrh_energy,  pfrh_topoId,  pfrh_layer, pfrh_depth, neigh8_Ind);
      }
     hipDeviceSynchronize();
    
    

      //dim3 grid( (size+32-1)/32, (size+32-1)/32 );
      //dim3 block( 32, 32);

      if(size>0) hcalFastCluster_step1_serialize<<<1,1>>>( size, pfrh_x,  pfrh_y,  pfrh_z,  pfrh_energy, pfrh_topoId,  pfrh_isSeed,  pfrh_layer, pfrh_depth, pcrhfrac, pcrhfracind, fracSum, rhCount);
     //hipDeviceSynchronize();

      if(size>0) hcalFastCluster_step2_serialize<<<1,1>>>( size, pfrh_x,  pfrh_y,  pfrh_z,  pfrh_energy, pfrh_topoId,  pfrh_isSeed,  pfrh_layer, pfrh_depth, pcrhfrac, pcrhfracind, fracSum, rhCount);


  }

void PFRechitToPFCluster_HCAL_serialize_step1Parallel(size_t size, 
				float* pfrh_x, 
				float* pfrh_y, 
				float* pfrh_z, 
				double* pfrh_energy, 
				double* pfrh_pt2,    				
				int* pfrh_isSeed,
				int* pfrh_topoId, 
				int* pfrh_layer, 
				int* pfrh_depth, 
				int* neigh8_Ind, 
				int* neigh4_Ind, 				
				
				int* pcrhfracind,
				float* pcrhfrac,
				float* fracSum,
				int* rhCount
				)
  { 
    //seeding
    if(size>0) seedingKernel_HCAL_serialize<<<1,1>>>( size,  pfrh_energy,   pfrh_pt2,   pfrh_isSeed,  pfrh_topoId,  pfrh_layer,pfrh_depth,  neigh4_Ind);
    //hipDeviceSynchronize();
    
    //topoclustering 
      
      //dim3 gridT( (size+64-1)/64, 1 );
      //dim3 blockT( 64, 8);
      for(int h=0;h<nTopoLoops; h++){
      if(size>0) topoKernel_HCAL_serialize<<<1,1>>>( size, pfrh_energy,  pfrh_topoId,  pfrh_layer, pfrh_depth, neigh8_Ind); 
      }
     hipDeviceSynchronize();
    
    

      dim3 grid( (size+32-1)/32, (size+32-1)/32 );
      dim3 block( 32, 32);

      if(size>0) hcalFastCluster_step1<<<grid, block>>>( size, pfrh_x,  pfrh_y,  pfrh_z,  pfrh_energy, pfrh_topoId,  pfrh_isSeed,  pfrh_layer, pfrh_depth, pcrhfrac, pcrhfracind, fracSum, rhCount);
     //hipDeviceSynchronize();

      if(size>0) hcalFastCluster_step2_serialize<<<1,1>>>( size, pfrh_x,  pfrh_y,  pfrh_z,  pfrh_energy, pfrh_topoId,  pfrh_isSeed,  pfrh_layer, pfrh_depth, pcrhfrac, pcrhfracind, fracSum, rhCount);


  }

void PFRechitToPFCluster_HCAL_serialize_step2Parallel(size_t size, 
				float* pfrh_x, 
				float* pfrh_y, 
				float* pfrh_z, 
				double* pfrh_energy, 
				double* pfrh_pt2,    				
				int* pfrh_isSeed,
				int* pfrh_topoId, 
				int* pfrh_layer, 
				int* pfrh_depth, 
				int* neigh8_Ind, 
				int* neigh4_Ind, 				
				
				int* pcrhfracind,
				float* pcrhfrac,
				float* fracSum,
				int* rhCount
				)
  { 
    //seeding
    if(size>0) seedingKernel_HCAL_serialize<<<1,1>>>( size,  pfrh_energy,   pfrh_pt2,   pfrh_isSeed,  pfrh_topoId,  pfrh_layer,pfrh_depth,  neigh4_Ind);
    //hipDeviceSynchronize();
    
    //topoclustering 
      
      //dim3 gridT( (size+64-1)/64, 1 );
      //dim3 blockT( 64, 8);
      for(int h=0;h<nTopoLoops; h++){
      if(size>0) topoKernel_HCAL_serialize<<<1,1>>>( size, pfrh_energy,  pfrh_topoId,  pfrh_layer, pfrh_depth, neigh8_Ind); 
      }
     hipDeviceSynchronize();
    
    

      dim3 grid( (size+32-1)/32, (size+32-1)/32 );
      dim3 block( 32, 32);

      if(size>0) hcalFastCluster_step1_serialize<<<1,1>>>( size, pfrh_x,  pfrh_y,  pfrh_z,  pfrh_energy, pfrh_topoId,  pfrh_isSeed,  pfrh_layer, pfrh_depth, pcrhfrac, pcrhfracind, fracSum, rhCount);
     //hipDeviceSynchronize();
      
      if(size>0) hcalFastCluster_step2<<<grid, block>>>( size, pfrh_x,  pfrh_y,  pfrh_z,  pfrh_energy, pfrh_topoId,  pfrh_isSeed,  pfrh_layer, pfrh_depth, pcrhfrac, pcrhfracind, fracSum, rhCount);

  }

} // namespace cudavectors
