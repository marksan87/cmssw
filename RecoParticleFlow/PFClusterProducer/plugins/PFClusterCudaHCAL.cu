#include "hip/hip_runtime.h"

#include <cmath>
#include <iostream>

// CUDA include files
#include <hip/hip_runtime.h>

// CMSSW include files
#include "HeterogeneousCore/CUDAUtilities/interface/cudaCheck.h"
#include "RecoParticleFlow/PFClusterProducer/plugins/PFClusterCudaHCAL.h"
#include <Eigen/Dense>
#include <hip/hip_runtime_api.h>

// Uncomment for debugging
#define DEBUG_GPU_HCAL

constexpr int sizeof_float = sizeof(float);
constexpr int sizeof_int = sizeof(int);


namespace PFClusterCudaHCAL {

  __constant__ float showerSigma2;
  __constant__ float recHitEnergyNormEB_vec[4];
  __constant__ float recHitEnergyNormEE_vec[7];
  __constant__ float minFracToKeep;
  __constant__ float minFracTot;
  __constant__ float stoppingTolerance;

  __constant__ float seedEThresholdEB_vec[4];
  __constant__ float seedEThresholdEE_vec[7];
  __constant__ float seedPt2ThresholdEB;
  __constant__ float seedPt2ThresholdEE;

  __constant__ float topoEThresholdEB_vec[4];
  __constant__ float topoEThresholdEE_vec[7];
 
  __constant__ int maxIterations;
  __constant__ bool excludeOtherSeeds;

  __constant__ int nNT = 8;  // Number of neighbors considered for topo clustering
  __constant__ int nNeigh;
  __constant__ int maxSize;
 
  //int nTopoLoops = 100;
  int nTopoLoops = 35;


  bool initializeCudaConstants(float h_showerSigma2,
                               const float (&h_recHitEnergyNormEB_vec)[4],
                               const float (&h_recHitEnergyNormEE_vec)[7],
                               float h_minFracToKeep,
                               float h_minFracTot,
                               int   h_maxIterations,
                               float h_stoppingTolerance,
                               bool  h_excludeOtherSeeds,
                               const float (&h_seedEThresholdEB_vec)[4],
                               const float (&h_seedEThresholdEE_vec)[7],
                               float h_seedPt2ThresholdEB,
                               float h_seedPt2ThresholdEE,
                               const float (&h_topoEThresholdEB_vec)[4],
                               const float (&h_topoEThresholdEE_vec)[7],
                               int   h_nNeigh,
                               int   h_maxSize
                           )
  {
     bool status = true;
     status &= cudaCheck(hipMemcpyToSymbolAsync(HIP_SYMBOL(showerSigma2), &h_showerSigma2, sizeof_float));
#ifdef DEBUG_GPU_HCAL
     // Read back the value
     std::cout<<"--- HCAL Cuda constant values ---"<<std::endl;
     float val = 0.;
     status &= cudaCheck(hipMemcpyFromSymbol(&val, HIP_SYMBOL(showerSigma2), sizeof_float));
     std::cout<<"showerSigma2 read from symbol: "<<val<<std::endl;
#endif
     
     status &= cudaCheck(hipMemcpyToSymbolAsync(HIP_SYMBOL(recHitEnergyNormEB_vec), &h_recHitEnergyNormEB_vec, 4*sizeof_float));
#ifdef DEBUG_GPU_HCAL
     // Read back the value
     float val4[4];
     status &= cudaCheck(hipMemcpyFromSymbol(&val4, HIP_SYMBOL(recHitEnergyNormEB_vec), 4*sizeof_float));
     std::cout<<"recHitEnergyNormEB_vec read from symbol: ";
     for (int i = 0; i < 4; i++) {std::cout<<val4[i]<<" ";}
     std::cout<<std::endl;
#endif

     status &= cudaCheck(hipMemcpyToSymbolAsync(HIP_SYMBOL(recHitEnergyNormEE_vec), &h_recHitEnergyNormEE_vec, 7*sizeof_float));
#ifdef DEBUG_GPU_HCAL
     // Read back the value
     float val7[7];
     status &= cudaCheck(hipMemcpyFromSymbol(&val7, HIP_SYMBOL(recHitEnergyNormEE_vec), 7*sizeof_float));
     std::cout<<"recHitEnergyNormEE_vec read from symbol: ";
     for (int i = 0; i < 7; i++) {std::cout<<val7[i]<<" ";}
     std::cout<<std::endl;
#endif

     status &= cudaCheck(hipMemcpyToSymbolAsync(HIP_SYMBOL(minFracToKeep), &h_minFracToKeep, sizeof_float));
#ifdef DEBUG_GPU_HCAL
     // Read back the value
     val = 0.;
     status &= cudaCheck(hipMemcpyFromSymbol(&val, HIP_SYMBOL(minFracToKeep), sizeof_float));
     std::cout<<"minFracToKeep read from symbol: "<<val<<std::endl;
#endif
    
     status &= cudaCheck(hipMemcpyToSymbolAsync(HIP_SYMBOL(minFracTot), &h_minFracTot, sizeof_float));
#ifdef DEBUG_GPU_ECAL
     // Read back the value
     val = 0.;
     status &= cudaCheck(hipMemcpyFromSymbol(&val, HIP_SYMBOL(minFracTot), sizeof_float));
     std::cout<<"minFracTot read from symbol: "<<val<<std::endl;
#endif

     status &= cudaCheck(hipMemcpyToSymbolAsync(HIP_SYMBOL(stoppingTolerance), &h_stoppingTolerance, sizeof_float));
#ifdef DEBUG_GPU_ECAL
     // Read back the value
     val = 0.;
     status &= cudaCheck(hipMemcpyFromSymbol(&val, HIP_SYMBOL(stoppingTolerance), sizeof_float));
     std::cout<<"stoppingTolerance read from symbol: "<<val<<std::endl;
#endif

     status &= cudaCheck(hipMemcpyToSymbolAsync(HIP_SYMBOL(excludeOtherSeeds), &h_excludeOtherSeeds, sizeof(bool)));
#ifdef DEBUG_GPU_ECAL
     // Read back the value
     bool bval = 0.;
     status &= cudaCheck(hipMemcpyFromSymbol(&bval, HIP_SYMBOL(excludeOtherSeeds), sizeof(bool)));
     std::cout<<"excludeOtherSeeds read from symbol: "<<bval<<std::endl;
#endif

     status &= cudaCheck(hipMemcpyToSymbolAsync(HIP_SYMBOL(maxIterations), &h_maxIterations, sizeof_int));
#ifdef DEBUG_GPU_ECAL
     // Read back the value
     int ival = 0.;
     status &= cudaCheck(hipMemcpyFromSymbol(&ival, HIP_SYMBOL(maxIterations), sizeof_int));
     std::cout<<"maxIterations read from symbol: "<<ival<<std::endl;
#endif

     status &= cudaCheck(hipMemcpyToSymbolAsync(HIP_SYMBOL(seedEThresholdEB_vec), &h_seedEThresholdEB_vec, 4*sizeof_float));
#ifdef DEBUG_GPU_HCAL
     // Read back the value
     status &= cudaCheck(hipMemcpyFromSymbol(&val4, HIP_SYMBOL(seedEThresholdEB_vec), 4*sizeof_float));
     std::cout<<"seedEThresholdEB_vec read from symbol: ";
     for (int i = 0; i < 4; i++) {std::cout<<val4[i]<<" ";}
     std::cout<<std::endl;
#endif

     status &= cudaCheck(hipMemcpyToSymbolAsync(HIP_SYMBOL(seedEThresholdEE_vec), &h_seedEThresholdEE_vec, 7*sizeof_float));
#ifdef DEBUG_GPU_HCAL
     // Read back the value
     status &= cudaCheck(hipMemcpyFromSymbol(&val7, HIP_SYMBOL(seedEThresholdEE_vec), 7*sizeof_float));
     std::cout<<"seedEThresholdEE_vec read from symbol: ";
     for (int i = 0; i < 7; i++) {std::cout<<val7[i]<<" ";}
     std::cout<<std::endl;
#endif

     status &= cudaCheck(hipMemcpyToSymbolAsync(HIP_SYMBOL(seedPt2ThresholdEB), &h_seedPt2ThresholdEB, sizeof_float));
#ifdef DEBUG_GPU_HCAL
     // Read back the value
     val = 0.;
     status &= cudaCheck(hipMemcpyFromSymbol(&val, HIP_SYMBOL(seedPt2ThresholdEB), sizeof_float));
     std::cout<<"seedPt2ThresholdEB read from symbol: "<<val<<std::endl;
#endif

     status &= cudaCheck(hipMemcpyToSymbolAsync(HIP_SYMBOL(seedPt2ThresholdEE), &h_seedPt2ThresholdEE, sizeof_float));
#ifdef DEBUG_GPU_HCAL
     // Read back the value
     val = 0.;
     status &= cudaCheck(hipMemcpyFromSymbol(&val, HIP_SYMBOL(seedPt2ThresholdEE), sizeof_float));
     std::cout<<"seedPt2ThresholdEE read from symbol: "<<val<<std::endl;
#endif

     status &= cudaCheck(hipMemcpyToSymbolAsync(HIP_SYMBOL(topoEThresholdEB_vec), &h_topoEThresholdEB_vec, 4*sizeof_float));
#ifdef DEBUG_GPU_HCAL
     // Read back the value
     status &= cudaCheck(hipMemcpyFromSymbol(&val4, HIP_SYMBOL(topoEThresholdEB_vec), 4*sizeof_float));
     std::cout<<"topoEThresholdEB_vec read from symbol: ";
     for (int i = 0; i < 4; i++) {std::cout<<val4[i]<<" ";}
     std::cout<<std::endl;
#endif

     status &= cudaCheck(hipMemcpyToSymbolAsync(HIP_SYMBOL(topoEThresholdEE_vec), &h_topoEThresholdEE_vec, 7*sizeof_float));
#ifdef DEBUG_GPU_HCAL
     // Read back the value
     status &= cudaCheck(hipMemcpyFromSymbol(&val7, HIP_SYMBOL(topoEThresholdEE_vec), 7*sizeof_float));
     std::cout<<"topoEThresholdEE_vec read from symbol: ";
     for (int i = 0; i < 7; i++) {std::cout<<val7[i]<<" ";}
     std::cout<<std::endl;
#endif

     status &= cudaCheck(hipMemcpyToSymbolAsync(HIP_SYMBOL(nNeigh), &h_nNeigh, sizeof_int));
#ifdef DEBUG_GPU_HCAL
     // Read back the value
     int ival = 0;
     status &= cudaCheck(hipMemcpyFromSymbol(&ival, HIP_SYMBOL(nNeigh), sizeof_int));
     std::cout<<"nNeigh read from symbol: "<<ival<<std::endl;
#endif

     status &= cudaCheck(hipMemcpyToSymbolAsync(HIP_SYMBOL(maxSize), &h_maxSize, sizeof_int));
#ifdef DEBUG_GPU_HCAL
     // Read back the value
     ival = 0;
     status &= cudaCheck(hipMemcpyFromSymbol(&ival, HIP_SYMBOL(maxSize), sizeof_int));
     std::cout<<"maxSize read from symbol: "<<ival<<std::endl;
#endif
     
     return status;
}
 
 __global__ void seedingTopoThreshKernel_HCAL(
     				size_t size, 
				    const double* __restrict__ pfrh_energy,
				    const double* __restrict__ pfrh_pt2,
				    int*   pfrh_isSeed,
				    int*   pfrh_topoId,
                    bool*  pfrh_passTopoThresh,
				    const int* __restrict__ pfrh_layer,
				    const int* __restrict__ pfrh_depth,
				    const int* __restrict__ neigh4_Ind
				    ) {

   int i = threadIdx.x+blockIdx.x*blockDim.x;

   if(i<size) {
     // Seeding threshold test
     if ( (pfrh_layer[i] == 1 && pfrh_energy[i]>seedEThresholdEB_vec[pfrh_depth[i]-1] && pfrh_pt2[i]>seedPt2ThresholdEB) || 
          (pfrh_layer[i] == 3 && pfrh_energy[i]>seedEThresholdEE_vec[pfrh_depth[i]-1] && pfrh_pt2[i]>seedPt2ThresholdEE) )
       {
	 pfrh_isSeed[i]=1;		 
	 for(int k=0; k<nNeigh; k++){
	   if(neigh4_Ind[nNeigh*i+k]<0) continue; 
	   if(pfrh_energy[i]<pfrh_energy[neigh4_Ind[nNeigh*i+k]]){
	     pfrh_isSeed[i]=0;
	     //pfrh_topoId[i]=-1;	     
	     break;
	   }
	 }		
       }
     else{ 
       // pfrh_topoId[i]=-1;
       pfrh_isSeed[i]=0;
       	    
     }
    
     // Topo clustering threshold test
     if ( (pfrh_layer[i] == 3 && pfrh_energy[i]>topoEThresholdEE_vec[pfrh_depth[i]-1]) ||
             (pfrh_layer[i] == 1 && pfrh_energy[i]>topoEThresholdEB_vec[pfrh_depth[i]-1])) {
            pfrh_passTopoThresh[i] = true;
        }
     else { pfrh_passTopoThresh[i] = false; }
   }
 }
 __global__ void seedingKernel_HCAL(
     				size_t size, 
				    const double* __restrict__ pfrh_energy,
				    const double* __restrict__ pfrh_pt2,
				    int*   pfrh_isSeed,
				    int*   pfrh_topoId,
				    const int* __restrict__ pfrh_layer,
				    const int* __restrict__ pfrh_depth,
				    const int* __restrict__ neigh4_Ind
				    ) {

   int i = threadIdx.x+blockIdx.x*blockDim.x;

   if(i<size) {        
     if ( (pfrh_layer[i] == 1 && pfrh_energy[i]>seedEThresholdEB_vec[pfrh_depth[i]-1] && pfrh_pt2[i]>seedPt2ThresholdEB) || 
          (pfrh_layer[i] == 3 && pfrh_energy[i]>seedEThresholdEE_vec[pfrh_depth[i]-1] && pfrh_pt2[i]>seedPt2ThresholdEE) )
       {
	 pfrh_isSeed[i]=1;		 
	 for(int k=0; k<nNeigh; k++){
	   if(neigh4_Ind[nNeigh*i+k]<0) continue; 
	   if(pfrh_energy[i]<pfrh_energy[neigh4_Ind[nNeigh*i+k]]){
	     pfrh_isSeed[i]=0;
	     //pfrh_topoId[i]=-1;	     
	     break;
	   }
	 }		
       }
     else{ 
       // pfrh_topoId[i]=-1;
       pfrh_isSeed[i]=0;
       	    
     }     
   }
 }

  
 __global__ void seedingKernel_HCAL_serialize(
     				    size_t size, 
				    const double* __restrict__ pfrh_energy,
				    const double* __restrict__ pfrh_pt2,
				    int*   pfrh_isSeed,
				    int*   pfrh_topoId,
				    const int* __restrict__ pfrh_layer,
				    const int* __restrict__ pfrh_depth,
				    const int* __restrict__ neigh4_Ind
				    ) {

   //int i = threadIdx.x+blockIdx.x*blockDim.x;
   for (int i = 0; i < size; i++) {
       if(i<size) {        
         if ( (pfrh_layer[i] == 1 && pfrh_energy[i]>seedEThresholdEB_vec[pfrh_depth[i]-1] && pfrh_pt2[i]>seedPt2ThresholdEB) || 
              (pfrh_layer[i] == 3 && pfrh_energy[i]>seedEThresholdEE_vec[pfrh_depth[i]-1] && pfrh_pt2[i]>seedPt2ThresholdEE) )
           {
         pfrh_isSeed[i]=1;		 
         for(int k=0; k<nNeigh; k++){
           if(neigh4_Ind[nNeigh*i+k]<0) continue; 
           if(pfrh_energy[i]<pfrh_energy[neigh4_Ind[nNeigh*i+k]]){
             pfrh_isSeed[i]=0;
             //pfrh_topoId[i]=-1;	     
             break;
           }
         }		
           }
         else{ 
           // pfrh_topoId[i]=-1;
           pfrh_isSeed[i]=0;
                
         }     
       }
    }
 }
 
 __global__ void topoKernel_HCAL_passTopoThresh(
    size_t size,
    const double* __restrict__ pfrh_energy,
    int* pfrh_topoId,
    const bool* __restrict__ pfrh_passTopoThresh,
    const int* __restrict__ neigh8_Ind
) {

    int l = threadIdx.x + blockIdx.x*blockDim.x;
    int k = (threadIdx.y + blockIdx.y*blockDim.y) % nNT;

    //if(l<size && k<nNT) {
    if (l < size) {

        while (pfrh_passTopoThresh[nNT*l + k] && neigh8_Ind[nNT*l + k] > -1 && pfrh_topoId[l] != pfrh_topoId[neigh8_Ind[nNT*l + k]])
        {
            if (pfrh_topoId[l] > pfrh_topoId[neigh8_Ind[nNT*l + k]]) {
                atomicMax(&pfrh_topoId[neigh8_Ind[nNT*l + k]], pfrh_topoId[l]);
            }
            if (pfrh_topoId[l] < pfrh_topoId[neigh8_Ind[nNT*l + k]]) {
                atomicMax(&pfrh_topoId[l], pfrh_topoId[neigh8_Ind[nNT*l + k]]);
            }
        }
    }
}

   __global__ void topoKernel_HCALV2( 
				  size_t size,
				  const double* __restrict__ pfrh_energy,
				  int* pfrh_topoId,
				  const int* __restrict__ pfrh_layer,
				  const int* __restrict__ pfrh_depth,
				  const int* __restrict__ neigh8_Ind
				  ) {
     
     int l = threadIdx.x+blockIdx.x*blockDim.x;
     //int k = threadIdx.y+blockIdx.y*blockDim.y;
     int k = (threadIdx.y+blockIdx.y*blockDim.y) % nNT;
           
      //if(l<size && k<nNT) {
      if(l<size) {

	while( neigh8_Ind[nNT*l+k] > -1 && pfrh_topoId[l] != pfrh_topoId[neigh8_Ind[nNT*l+k]] && 
	    ( (pfrh_layer[neigh8_Ind[nNT*l+k]] == 3 && pfrh_energy[neigh8_Ind[nNT*l+k]]>topoEThresholdEE_vec[pfrh_depth[neigh8_Ind[nNT*l+k]]-1]) ||
	      (pfrh_layer[neigh8_Ind[nNT*l+k]] == 1 && pfrh_energy[neigh8_Ind[nNT*l+k]]>topoEThresholdEB_vec[pfrh_depth[neigh8_Ind[nNT*l+k]]-1]) ) &&
	    ( (pfrh_layer[l] == 3 && pfrh_energy[l]>topoEThresholdEE_vec[pfrh_depth[l]-1]) ||
	      (pfrh_layer[l] == 1 && pfrh_energy[l]>topoEThresholdEB_vec[pfrh_depth[l]-1]))
	    )
	    {
	      if(pfrh_topoId[l] > pfrh_topoId[neigh8_Ind[nNT*l+k]]){
		atomicMax(&pfrh_topoId[neigh8_Ind[nNT*l+k]],pfrh_topoId[l]);
	      }
	      if(pfrh_topoId[l] < pfrh_topoId[neigh8_Ind[nNT*l+k]]){
		atomicMax(&pfrh_topoId[l], pfrh_topoId[neigh8_Ind[nNT*l+k]]);
	      }
	    }	
      }
   }
 

   __global__ void topoKernel_HCAL_serialize( 
				  size_t size,
				  const double* __restrict__ pfrh_energy,
				  int* pfrh_topoId,
				  const int* __restrict__ pfrh_layer,
				  const int* __restrict__ pfrh_depth,
				  const int* __restrict__ neigh8_Ind
				  ) {
     
     //int l = threadIdx.x+blockIdx.x*blockDim.x;
     //int k = threadIdx.y+blockIdx.y*blockDim.y;
     
     for (int l = 0; l < size; l++) {
        //for (int k = 0; k < size; k++) {
        for (int k = 0; k < 8; k++) {
           
            while( neigh8_Ind[nNT*l+k] > -1 && pfrh_topoId[l] != pfrh_topoId[neigh8_Ind[nNT*l+k]] && 
                ( (pfrh_layer[neigh8_Ind[nNT*l+k]] == 3 && pfrh_energy[neigh8_Ind[nNT*l+k]]>topoEThresholdEE_vec[pfrh_depth[neigh8_Ind[nNT*l+k]]-1]) ||
                  (pfrh_layer[neigh8_Ind[nNT*l+k]] == 1 && pfrh_energy[neigh8_Ind[nNT*l+k]]>topoEThresholdEB_vec[pfrh_depth[neigh8_Ind[nNT*l+k]]-1]) ) &&
                ( (pfrh_layer[l] == 3 && pfrh_energy[l]>topoEThresholdEE_vec[pfrh_depth[l]-1]) ||
                  (pfrh_layer[l] == 1 && pfrh_energy[l]>topoEThresholdEB_vec[pfrh_depth[l]-1]))
                )
                {
                  if(pfrh_topoId[l] > pfrh_topoId[neigh8_Ind[nNT*l+k]]){
                atomicMax(&pfrh_topoId[neigh8_Ind[nNT*l+k]],pfrh_topoId[l]);
                  }
                  if(pfrh_topoId[l] < pfrh_topoId[neigh8_Ind[nNT*l+k]]){
                atomicMax(&pfrh_topoId[l], pfrh_topoId[neigh8_Ind[nNT*l+k]]);
                  }
                }	
          }
        }
   }
 
__global__ void hcalFastCluster_step1( size_t size,
					     const float* __restrict__ pfrh_x,
					     const float* __restrict__ pfrh_y,
					     const float* __restrict__ pfrh_z,
					     const double* __restrict__ pfrh_energy,
					     int* pfrh_topoId,
					     int* pfrh_isSeed,
					     const int* __restrict__ pfrh_layer,
				         const int* __restrict__ pfrh_depth,
					     float* pcrhfrac,
					     int* pcrhfracind,
					     float* fracSum,
					     int* rhCount
					     ) {

    int i = threadIdx.x+blockIdx.x*blockDim.x;
    int j = threadIdx.y+blockIdx.y*blockDim.y;
    //make sure topoID, Layer is the same, i is seed and j is not seed
    if( i<size && j<size){

      if( pfrh_topoId[i] == pfrh_topoId[j] && pfrh_isSeed[i]==1 ){

      float dist2 =
	       (pfrh_x[i] - pfrh_x[j])*(pfrh_x[i] - pfrh_x[j])
	      +(pfrh_y[i] - pfrh_y[j])*(pfrh_y[i] - pfrh_y[j])
	      +(pfrh_z[i] - pfrh_z[j])*(pfrh_z[i] - pfrh_z[j]);

      float d2 = dist2 / showerSigma2;
      float fraction = -1.;

      if(pfrh_layer[j] == 1) { fraction = pfrh_energy[i] / recHitEnergyNormEB_vec[pfrh_depth[j] - 1] * expf(-0.5 * d2); }
      else if (pfrh_layer[j] == 3) { fraction = pfrh_energy[i] / recHitEnergyNormEE_vec[pfrh_depth[j] - 1] * expf(-0.5 * d2); }
	  
      if(fraction==-1.) printf("FRACTION is NEGATIVE!!!");

      if( pfrh_isSeed[j]!=1 )
	{
	  atomicAdd(&fracSum[j],fraction);
	}
      }
    }
}


__global__ void hcalFastCluster_step2( size_t size,
					     const float* __restrict__ pfrh_x,
					     const float* __restrict__ pfrh_y,
					     const float* __restrict__ pfrh_z,
					     const double* __restrict__ pfrh_energy,
					     int* pfrh_topoId,
					     int* pfrh_isSeed,
					     const int* __restrict__ pfrh_layer,
				         const int* __restrict__ pfrh_depth,
					     float* pcrhfrac,
					     int* pcrhfracind,
					     float* fracSum,
					     int* rhCount
					     ) {

    int i = threadIdx.x+blockIdx.x*blockDim.x;
    int j = threadIdx.y+blockIdx.y*blockDim.y;
    //make sure topoID, Layer is the same, i is seed and j is not seed
    if( i<size && j<size){
      if( pfrh_topoId[i] == pfrh_topoId[j] && pfrh_isSeed[i]==1 ){
      if(i==j)
	{
	  pcrhfrac[i*maxSize]    = 1.;
	  pcrhfracind[i*maxSize] = j;
	}
      if( pfrh_isSeed[j]!=1 ){
        float dist2 =
           (pfrh_x[i] - pfrh_x[j])*(pfrh_x[i] - pfrh_x[j])
          +(pfrh_y[i] - pfrh_y[j])*(pfrh_y[i] - pfrh_y[j])
          +(pfrh_z[i] - pfrh_z[j])*(pfrh_z[i] - pfrh_z[j]);

        float d2 = dist2 / showerSigma2; 
        float fraction = -1.;

        if(pfrh_layer[j] == 1) { fraction = pfrh_energy[i] / recHitEnergyNormEB_vec[pfrh_depth[j] - 1] * expf(-0.5 * d2); }
        else if (pfrh_layer[j] == 3) { fraction = pfrh_energy[i] / recHitEnergyNormEE_vec[pfrh_depth[j] - 1] * expf(-0.5 * d2); }
        
        if(fraction==-1.) printf("FRACTION is NEGATIVE!!!");
        
        if (fracSum[j] > minFracTot) {
            float fracpct = fraction / fracSum[j];
            if(fracpct > 0.9999 || (d2 < 100. && fracpct > minFracToKeep))
              {
                  int k = atomicAdd(&rhCount[i],1);
                  pcrhfrac[i*maxSize+k] = fracpct;
                  pcrhfracind[i*maxSize+k] = j;
              }
        }
        /*
        if(d2 < 100. )
          {
            if ((fraction/fracSum[j])>minFracToKeep){
              int k = atomicAdd(&rhCount[i],1);
              pcrhfrac[i*maxSize+k] = fraction/fracSum[j];
              pcrhfracind[i*maxSize+k] = j;
              //printf("(i,j)=(%d,%d), rhCount=%d, fraction=%f, fracsum=%f \n",i,j,rhCount[i], fraction, fracSum[j]);
            }
          }
        */
      }
      }
    }
}


__global__ void hcalFastCluster_step1_serialize( size_t size,
					     const float* __restrict__ pfrh_x,
					     const float* __restrict__ pfrh_y,
					     const float* __restrict__ pfrh_z,
					     const double* __restrict__ pfrh_energy,
					     int* pfrh_topoId,
					     int* pfrh_isSeed,
					     const int* __restrict__ pfrh_layer,
				         const int* __restrict__ pfrh_depth,
					     float* pcrhfrac,
					     int* pcrhfracind,
					     float* fracSum,
					     int* rhCount
					     ) {

    //int i = threadIdx.x+blockIdx.x*blockDim.x;
    //int j = threadIdx.y+blockIdx.y*blockDim.y;
    for (int i = 0; i < size; i++) {
        for (int j = 0; j < size; j++) {
            //make sure topoID, Layer is the same, i is seed and j is not seed
            if( i<size && j<size){

              if( pfrh_topoId[i] == pfrh_topoId[j] && pfrh_isSeed[i]==1 ){

              float dist2 =
                   (pfrh_x[i] - pfrh_x[j])*(pfrh_x[i] - pfrh_x[j])
                  +(pfrh_y[i] - pfrh_y[j])*(pfrh_y[i] - pfrh_y[j])
                  +(pfrh_z[i] - pfrh_z[j])*(pfrh_z[i] - pfrh_z[j]);

              float d2 = dist2 / showerSigma2; 
              float fraction = -1.;

              if(pfrh_layer[j] == 1) { fraction = pfrh_energy[i] / recHitEnergyNormEB_vec[pfrh_depth[j] - 1] * expf(-0.5 * d2); }
              else if (pfrh_layer[j] == 3) { fraction = pfrh_energy[i] / recHitEnergyNormEE_vec[pfrh_depth[j] - 1] * expf(-0.5 * d2); }
              
              if(fraction==-1.) printf("FRACTION is NEGATIVE!!!");

              if( pfrh_isSeed[j]!=1 )
            {
              atomicAdd(&fracSum[j],fraction);
            }
              }
            }
        }
    }
  }



__global__ void hcalFastCluster_step2_serialize( size_t size,
					     const float* __restrict__ pfrh_x,
					     const float* __restrict__ pfrh_y,
					     const float* __restrict__ pfrh_z,
					     const double* __restrict__ pfrh_energy,
					     int* pfrh_topoId,
					     int* pfrh_isSeed,
					     const int* __restrict__ pfrh_layer,
				         const int* __restrict__ pfrh_depth,
					     float* pcrhfrac,
					     int* pcrhfracind,
					     float* fracSum,
					     int* rhCount
					     ) {

    //int i = threadIdx.x+blockIdx.x*blockDim.x;
    //int j = threadIdx.y+blockIdx.y*blockDim.y;
    //make sure topoID, Layer is the same, i is seed and j is not seed
    for (int i = 0; i < size; i++) {
        for (int j = 0; j < size; j++) {
            if( i<size && j<size){
              if( pfrh_topoId[i] == pfrh_topoId[j] && pfrh_isSeed[i]==1 ){
              if(i==j)
            {
              pcrhfrac[i*maxSize]    = 1.;
              pcrhfracind[i*maxSize] = j;
            }
              if( pfrh_isSeed[j]!=1 ){
            float dist2 =
               (pfrh_x[i] - pfrh_x[j])*(pfrh_x[i] - pfrh_x[j])
              +(pfrh_y[i] - pfrh_y[j])*(pfrh_y[i] - pfrh_y[j])
              +(pfrh_z[i] - pfrh_z[j])*(pfrh_z[i] - pfrh_z[j]);

            float d2 = dist2 / showerSigma2; 
            float fraction = -1.;

            if(pfrh_layer[j] == 1) { fraction = pfrh_energy[i] / recHitEnergyNormEB_vec[pfrh_depth[j] - 1] * expf(-0.5 * d2); }
            else if (pfrh_layer[j] == 3) { fraction = pfrh_energy[i] / recHitEnergyNormEE_vec[pfrh_depth[j] - 1] * expf(-0.5 * d2); }
              
            
            if(fraction==-1.) printf("FRACTION is NEGATIVE!!!");
            if(d2 < 100. )
              {
                if ((fraction/fracSum[j])>minFracToKeep){
                  int k = atomicAdd(&rhCount[i],1);
                  pcrhfrac[i*maxSize+k] = fraction/fracSum[j];
                  pcrhfracind[i*maxSize+k] = j;
                  //printf("(i,j)=(%d,%d), rhCount=%d, fraction=%f, fracsum=%f \n",i,j,rhCount[i], fraction, fracSum[j]);
                }
              }
              }
              }
            }
        }
    }
}

// Compute whether rechits pass topo clustering energy threshold
__global__ void passingTopoThreshold(size_t size,
                const int* __restrict__ pfrh_layer,
                const int* __restrict__ pfrh_depth,
                const double* __restrict__ pfrh_energy,
                bool* pfrh_passTopoThresh) {

    int i = threadIdx.x+blockIdx.x*blockDim.x;
    if (i < size) {
        if ( (pfrh_layer[i] == 3 && pfrh_energy[i]>topoEThresholdEE_vec[pfrh_depth[i]-1]) ||
             (pfrh_layer[i] == 1 && pfrh_energy[i]>topoEThresholdEB_vec[pfrh_depth[i]-1])) {
            pfrh_passTopoThresh[i] = true;
        }
        else { pfrh_passTopoThresh[i] = false; }
    }
}

__global__ void passingTopoThreshold(int size,
                const int* __restrict__ pfrh_layer,
                const int* __restrict__ pfrh_depth,
                const double* __restrict__ pfrh_energy,
                bool* pfrh_passTopoThresh) {

    int i = threadIdx.x+blockIdx.x*blockDim.x;
    if (i < size) {
        if ( (pfrh_layer[i] == 3 && pfrh_energy[i]>topoEThresholdEE_vec[pfrh_depth[i]-1]) ||
             (pfrh_layer[i] == 1 && pfrh_energy[i]>topoEThresholdEB_vec[pfrh_depth[i]-1])) {
            pfrh_passTopoThresh[i] = true;
        }
        else { pfrh_passTopoThresh[i] = false; }
    }
}

// Contraction in a single block
__global__ void topoClusterContraction(size_t size, int* pfrh_parent) {
    __shared__ int notDone;
    if (threadIdx.x == 0) notDone = 0;
    __syncthreads();

    do {
        volatile bool threadNotDone = false;
        for (int i = threadIdx.x; i < size; i += blockDim.x) {
            int parent = pfrh_parent[i];
            if (parent >= 0 && parent != pfrh_parent[parent]) {
                threadNotDone = true;
                pfrh_parent[i] = pfrh_parent[parent];
            }
        }
        if (threadIdx.x == 0) notDone = 0;
        __syncthreads();
        
        atomicAdd(&notDone, (int)threadNotDone);
        //if (threadNotDone) notDone = true;
        //notDone |= threadNotDone;
        __syncthreads();

    } while (notDone);
}


__device__ bool isLeftEdge(const int idx,
    const int nEdges,
    const int* __restrict__ pfrh_edgeId,
    const int* __restrict__ pfrh_edgeMask) {

    if (idx > 0) { 
        int temp = idx - 1;
        int minVal = max(idx - 9, 0);   //  Only test up to 9 neighbors
        int tempId = 0;
        int edgeId = pfrh_edgeId[idx];
        //int minVal = 0;
        while (temp >= minVal) {
            tempId = pfrh_edgeId[temp];
            if (edgeId != tempId) {
                // Different topo Id here!
                return true;
            }
            else if (pfrh_edgeMask[temp] > 0) {
                // Found adjacent edge
                return false;
            }
            temp--;
        }
    }
    else if (idx == 0) {
        return true;
    }

    // Invalid index
    return false;
}

__device__ bool isRightEdge(const int idx,
    const int nEdges,
    const int* __restrict__ pfrh_edgeId,
    const int* __restrict__ pfrh_edgeMask) {

    // Update right
    if (idx < (nEdges - 1)) {
        int temp = idx + 1;
        int maxVal = min(idx - 9, nEdges - 1);  //  Only test up to 9 neighbors
        //int maxVal = nEdges - 1;
        int tempId = 0;
        int edgeId = pfrh_edgeId[idx];
        while (temp >= maxVal) {
            tempId = pfrh_edgeId[temp];
            if (edgeId != tempId) {
                // Different topo Id here!
                return true;
            }
            else if (pfrh_edgeMask[temp] > 0) {
                // Found adjacent edge
                return false;
            }
            temp++;
        }
    }
    else if (idx == (nEdges - 1)) {
        return true;
    }

    // Overflow
    return false;
}


__global__ void topoClusterLinking(int nRH,
    int nEdges,
    int* pfrh_parent,
    int* pfrh_edgeId,
    int* pfrh_edgeList,
    int* pfrh_edgeMask,
    bool* pfrh_passTopoThresh,
    int* nIter) {

    __shared__ bool notDone;
    __shared__ int iter, gridStride;

    int start = blockIdx.x * blockDim.x + threadIdx.x;

    if (threadIdx.x == 0) {
        *nIter = 0;
        iter = 0;
        gridStride = blockDim.x * gridDim.x; // For single block kernel this is the number of threads
    }
    __syncthreads();

    // Check if pairs in edgeId,edgeList contain a rh not passing topo threshold
    // If found, set the mask to 0
    for (int idx = start; idx < nEdges; idx += gridStride) {
        if (pfrh_passTopoThresh[pfrh_edgeId[idx]] && pfrh_passTopoThresh[pfrh_edgeList[idx]])
            pfrh_edgeMask[idx] = 1;
        else
            pfrh_edgeMask[idx] = 0;
    }

    do {
        if (threadIdx.x == 0) {
            notDone = false;
        }
        __syncthreads();

        // Odd linking
        for (int idx = start; idx < nEdges; idx += gridStride) {
            int i = pfrh_edgeId[idx];   // Get edge topo id
            //if (pfrh_edgeMask[idx] > 0 && pfrh_passTopoThresh[i] && isLeftEdge(idx, nEdges, pfrh_edgeId, pfrh_edgeMask)) {
            if (pfrh_edgeMask[idx] > 0 && isLeftEdge(idx, nEdges, pfrh_edgeId, pfrh_edgeMask)) {
                pfrh_parent[i] = (int)min(i, pfrh_edgeList[idx]);
            }
        }

        __syncthreads();

        // edgeParent
        for (int idx = start; idx < nEdges; idx += gridStride) {
            if (pfrh_edgeMask[idx] > 0) {
                int id = pfrh_edgeId[idx];   // Get edge topo id
                int neighbor = pfrh_edgeList[idx]; // Get neighbor topo id
                pfrh_edgeId[idx] = pfrh_parent[id];
                pfrh_edgeList[idx] = pfrh_parent[neighbor];

                // edgeMask set to true if elements of edgeId and edgeList are different
                if (pfrh_edgeId[idx] != pfrh_edgeList[idx]) {
                    pfrh_edgeMask[idx] = 1;
                    notDone = true;
                }
                else {
                    pfrh_edgeMask[idx] = 0;
                }
            }
        }
        if (threadIdx.x == 0)
            iter++;

        __syncthreads();

        if (!notDone) break;

        if (threadIdx.x == 0) {
            notDone = false;
        }

        __syncthreads();

        // Even linking
        for (int idx = start; idx < nEdges; idx += gridStride) {
            int i = pfrh_edgeId[idx];   // Get edge topo id
            //if (pfrh_edgeMask[idx] > 0 && pfrh_passTopoThresh[i] && isRightEdge(idx, nEdges, pfrh_edgeId, pfrh_edgeMask)) {
            if (pfrh_edgeMask[idx] > 0 && isRightEdge(idx, nEdges, pfrh_edgeId, pfrh_edgeMask)) {
                pfrh_parent[i] = (int)max(i, pfrh_edgeList[idx]);
            }
        }

        __syncthreads();

        // edgeParent
        for (int idx = start; idx < nEdges; idx += gridStride) {
            if (pfrh_edgeMask[idx] > 0) {
                int id = pfrh_edgeId[idx];   // Get edge topo id
                int neighbor = pfrh_edgeList[idx]; // Get neighbor topo id
                pfrh_edgeId[idx] = pfrh_parent[id];
                pfrh_edgeList[idx] = pfrh_parent[neighbor];

                // edgeMask set to true if elements of edgeId and edgeList are different
                if (pfrh_edgeId[idx] != pfrh_edgeList[idx]) {
                    pfrh_edgeMask[idx] = 1;
                    notDone = true;
                }
                else {
                    pfrh_edgeMask[idx] = 0;
                }
            }
        }
        if (threadIdx.x == 0)
            iter++;

        __syncthreads();

    } while (notDone);
    *nIter = iter;
}


void PFRechitToPFCluster_HCAL_CCLClustering(int nRH,
                int nEdges,
                const float* __restrict__ pfrh_x,
                const float* __restrict__ pfrh_y,
                const float* __restrict__ pfrh_z,
                const double* __restrict__ pfrh_energy,
                const double* __restrict__ pfrh_pt2,
                int* pfrh_isSeed,
                int* pfrh_topoId,
                const int* __restrict__ pfrh_layer,
                const int* __restrict__ pfrh_depth,
                const int* __restrict__ neigh8_Ind,
                const int* __restrict__ neigh4_Ind,
                int* pfrh_edgeId,
                int* pfrh_edgeList,
                int* pfrh_edgeMask,
                bool* pfrh_passTopoThresh,
                int* pcrhfracind,
				float* pcrhfrac,
				float* fracSum,
                int* rhCount,
                float (&timer)[8],
                int* nIter) {
    if (nRH < 1) return;

#ifdef DEBUG_GPU_HCAL
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipDeviceSynchronize();
    hipEventRecord(start);
#endif
    hipProfilerStart();
    // Combined seeding & topo clustering thresholds
    seedingTopoThreshKernel_HCAL<<<(nRH+63/64), 128>>>(nRH, pfrh_energy, pfrh_pt2, pfrh_isSeed, pfrh_topoId, pfrh_passTopoThresh, pfrh_layer, pfrh_depth, neigh4_Ind);

#ifdef DEBUG_GPU_HCAL
    hipEventRecord(stop);
    hipEventSynchronize(stop);   
    hipEventElapsedTime(&timer[0], start, stop);
    hipDeviceSynchronize();
    hipEventRecord(start);
#endif
    
    //topoclustering 
    topoClusterLinking<<<1, 1024 >>>(nRH, nEdges, pfrh_topoId, pfrh_edgeId, pfrh_edgeList, pfrh_edgeMask, pfrh_passTopoThresh, nIter);
    topoClusterContraction<<<1, 512>>>(nRH, pfrh_topoId);

#ifdef DEBUG_GPU_HCAL
    hipEventRecord(stop);
    hipEventSynchronize(stop);   
    hipEventElapsedTime(&timer[1], start, stop);
    hipDeviceSynchronize();
    hipEventRecord(start);
#endif

    dim3 grid( (nRH+32-1)/32, (nRH+32-1)/32 );
    dim3 block( 32, 32);

    hcalFastCluster_step1<<<grid, block>>>( nRH, pfrh_x,  pfrh_y,  pfrh_z,  pfrh_energy, pfrh_topoId,  pfrh_isSeed,  pfrh_layer, pfrh_depth, pcrhfrac, pcrhfracind, fracSum, rhCount);

#ifdef DEBUG_GPU_HCAL
    hipEventRecord(stop);
    hipEventSynchronize(stop);   
    hipEventElapsedTime(&timer[2], start, stop);
    hipDeviceSynchronize();
    hipEventRecord(start);
#endif

    hcalFastCluster_step2<<<grid, block>>>( nRH, pfrh_x,  pfrh_y,  pfrh_z,  pfrh_energy, pfrh_topoId,  pfrh_isSeed,  pfrh_layer, pfrh_depth, pcrhfrac, pcrhfracind, fracSum, rhCount);

#ifdef DEBUG_GPU_HCAL
    hipEventRecord(stop);
    hipEventSynchronize(stop);   
    hipEventElapsedTime(&timer[3], start, stop);
    hipDeviceSynchronize();
#endif
    hipProfilerStop();
}


void PFRechitToPFCluster_HCALV2(size_t size, 
				const float* __restrict__ pfrh_x, 
				const float* __restrict__ pfrh_y, 
				const float* __restrict__ pfrh_z, 
				const double* __restrict__ pfrh_energy, 
				const double* __restrict__ pfrh_pt2,    				
				int* pfrh_isSeed,
				bool* pfrh_passTopoThresh,
				int* pfrh_topoId, 
				const int* __restrict__ pfrh_layer, 
				const int* __restrict__ pfrh_depth, 
			    const int* __restrict__ neigh8_Ind,
                const int* __restrict__ neigh4_Ind,	
				int* pcrhfracind,
				float* pcrhfrac,
				float* fracSum,
				int* rhCount,
				float (&timer)[8]
                )
  {
    if (size <= 0) return;
#ifdef DEBUG_GPU_HCAL
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
#endif
    hipProfilerStart();
    //seeding
    seedingKernel_HCAL<<<(size+512-1)/512, 512>>>( size,  pfrh_energy,   pfrh_pt2,   pfrh_isSeed,  pfrh_topoId,  pfrh_layer,pfrh_depth,  neigh4_Ind);

    // Passing topo clustering threshold
    passingTopoThreshold<<<(size+255)/256, 256>>>( size, pfrh_layer, pfrh_depth, pfrh_energy, pfrh_passTopoThresh);
#ifdef DEBUG_GPU_HCAL
      hipEventRecord(stop);
      hipEventSynchronize(stop);   
      hipEventElapsedTime(&timer[0], start, stop);
      hipEventRecord(start);
#endif
    
    //topoclustering 
     
      //hipProfilerStart();
      //dim3 gridT( (size+64-1)/64, 1 );
      //dim3 blockT( 64, 8);
      dim3 gridT( (size+64-1)/64, 8 );
      dim3 blockT( 64, 16); // 16 threads in a half-warp
      for(int h=0;h<nTopoLoops; h++){
        topoKernel_HCAL_passTopoThresh <<<gridT, blockT >>> (size, pfrh_energy, pfrh_topoId, pfrh_passTopoThresh, neigh8_Ind);
        //topoKernel_HCALV2<<<gridT, blockT>>>( size, pfrh_energy,  pfrh_topoId,  pfrh_layer, pfrh_depth, neigh8_Ind);	     
      }
   
      //hipProfilerStop();
#ifdef DEBUG_GPU_HCAL
      hipEventRecord(stop);
      hipEventSynchronize(stop);   
      hipEventElapsedTime(&timer[1], start, stop);
      hipEventRecord(start);
#endif

      dim3 grid( (size+32-1)/32, (size+32-1)/32 );
      dim3 block( 32, 32);

      hcalFastCluster_step1<<<grid, block>>>( size, pfrh_x,  pfrh_y,  pfrh_z,  pfrh_energy, pfrh_topoId,  pfrh_isSeed,  pfrh_layer, pfrh_depth, pcrhfrac, pcrhfracind, fracSum, rhCount);

#ifdef DEBUG_GPU_HCAL
      hipEventRecord(stop);
      hipEventSynchronize(stop);   
      hipEventElapsedTime(&timer[2], start, stop);
      hipEventRecord(start);
#endif

      hcalFastCluster_step2<<<grid, block>>>( size, pfrh_x,  pfrh_y,  pfrh_z,  pfrh_energy, pfrh_topoId,  pfrh_isSeed,  pfrh_layer, pfrh_depth, pcrhfrac, pcrhfracind, fracSum, rhCount);

#ifdef DEBUG_GPU_HCAL
      hipEventRecord(stop);
      hipEventSynchronize(stop);   
      hipEventElapsedTime(&timer[3], start, stop);
#endif

      hipProfilerStop();
  }

void PFRechitToPFCluster_HCAL_serialize(size_t size, 
				const float* __restrict__ pfrh_x, 
				const float* __restrict__ pfrh_y, 
				const float* __restrict__ pfrh_z, 
				const double* __restrict__ pfrh_energy, 
				const double* __restrict__ pfrh_pt2,    				
				int* pfrh_isSeed,
				int* pfrh_topoId, 
				const int* __restrict__ pfrh_layer, 
				const int* __restrict__ pfrh_depth, 
			    const int* __restrict__ neigh8_Ind,
                const int* __restrict__ neigh4_Ind,	
				int* pcrhfracind,
				float* pcrhfrac,
				float* fracSum,
				int* rhCount,
				float* timer
				)
  {
#ifdef DEBUG_GPU_HCAL
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
#endif
    //seeding
    if(size>0) seedingKernel_HCAL_serialize<<<1,1>>>( size,  pfrh_energy,   pfrh_pt2,   pfrh_isSeed,  pfrh_topoId,  pfrh_layer,pfrh_depth,  neigh4_Ind);
    
    //topoclustering 
      
#ifdef DEBUG_GPU_HCAL
      hipEventRecord(start);
#endif
      for(int h=0;h<nTopoLoops; h++){
    
      if(size>0) topoKernel_HCAL_serialize<<<1,1>>>( size, pfrh_energy,  pfrh_topoId,  pfrh_layer, pfrh_depth, neigh8_Ind);	     
      }
#ifdef DEBUG_GPU_HCAL
      float milliseconds = 0;
      if (timer != nullptr)
      {
          hipEventRecord(stop);
          hipEventSynchronize(stop);
          hipEventElapsedTime(&milliseconds, start, stop);
          *timer = milliseconds;
      }
#endif    

      if(size>0) hcalFastCluster_step1_serialize<<<1,1>>>( size, pfrh_x,  pfrh_y,  pfrh_z,  pfrh_energy, pfrh_topoId,  pfrh_isSeed,  pfrh_layer, pfrh_depth, pcrhfrac, pcrhfracind, fracSum, rhCount);

      if(size>0) hcalFastCluster_step2_serialize<<<1,1>>>( size, pfrh_x,  pfrh_y,  pfrh_z,  pfrh_energy, pfrh_topoId,  pfrh_isSeed,  pfrh_layer, pfrh_depth, pcrhfrac, pcrhfracind, fracSum, rhCount);


  }

} // namespace cudavectors
